#include "hip/hip_runtime.h"
__device__ inline void dropPP(int ix, const int ixsh, const int iz, const int it, ftype* RPchannel, const ftype& val) { 
  ix = (ix+Ns*NDT)%(Ns*NDT);
  if(drop_cells[ix*Nwarps+iz/WSIZE] >> iz%WSIZE &1){
    int channel_shift=0;
    for(int xprev=0; xprev<ixsh; xprev++) for(int iwarp=0; iwarp<Nwarps; iwarp++) channel_shift+= __popc(drop_cells[(ix+Ns*NDT-ixsh+xprev)%(Ns*NDT)*Nwarps+iwarp]);
    for(int iwarp=0; iwarp<iz/WSIZE; iwarp++) channel_shift+= __popc(drop_cells[ix*Nwarps+iwarp]);
    channel_shift+= __popc(drop_cells[ix*Nwarps+iz/WSIZE]<<(32-iz%WSIZE));
    RPchannel[channel_shift] = val;
    //else              RPchannel[channel_shift] = 0.5625*val+0.5625*valP-0.0625*valM-0.0625*valPP;
  }
}
