#include "hip/hip_runtime.h"
#include <mpi.h>

#define CHECK_ERROR(err) CheckError( err, __FILE__,__LINE__)
static void CheckError( hipError_t err, const char *file, int line) {
  if(err!=hipSuccess){
    fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

__global__ void kernel(float* p){
  printf("%d   %g\n",threadIdx.x, p[threadIdx.x]);
}

int main(int argc, char** argv){
  printf("mpi_init\n");
  MPI_Init (&argc, &argv);
  int rank, size;
  MPI_Comm_rank (MPI_COMM_WORLD, &rank);
  MPI_Comm_size (MPI_COMM_WORLD, &size);
  float* p1, *p2, *p3, *p4;
  if(rank==0) {
  printf("rank 0 malloc\n");
    CHECK_ERROR( hipSetDevice(0) );
    CHECK_ERROR( hipMalloc((void**)&p1, 100*sizeof(float)) );
    CHECK_ERROR( hipSetDevice(1) );
    CHECK_ERROR( hipMalloc((void**)&p2, 100*sizeof(float)) );
    for(int i=0;i<100; i++) { p1[i]=i; }
    for(int i=0;i<100; i++) { p2[i]=i+1000; }
  printf("rank 0 malloc ok\n");
  }
  if(rank==1) {
  printf("rank 1 malloc\n");
    CHECK_ERROR( hipSetDevice(0) );
    CHECK_ERROR( hipMalloc((void**)&p3, 100*sizeof(float)));
    CHECK_ERROR( hipSetDevice(1) );
    CHECK_ERROR( hipMalloc((void**)&p4, 100*sizeof(float)));
  printf("rank 1 malloc ok\n");
  }
  MPI_Request request1, request2;
  MPI_Status status;
  printf("start sendrev 1\n");
  if(rank==1) MPI_Irecv(p1, 100, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &request1);
  if(rank==0) MPI_Isend(p4, 100, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &request2);
  MPI_Wait(&request1, &status);
  MPI_Wait(&request2, &status);

  printf("start sendrev 2\n");
  if(rank==1) MPI_Irecv(p2, 100, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &request1);
  if(rank==0) MPI_Isend(p3, 100, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &request2);
  MPI_Wait(&request1, &status);
  MPI_Wait(&request2, &status);

  printf("device0\n");
  CHECK_ERROR( hipSetDevice(0) );
  kernel<<<1,100>>>(p3);
  printf("device1\n");
  CHECK_ERROR( hipSetDevice(1) );
  kernel<<<1,100>>>(p4);
  
  MPI_Finalize();

  return 0;

}
