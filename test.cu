
#include <hip/hip_runtime.h>
__global__ void kernel(float4* a, const hipTextureObject_t* tex){
  a[0] = tex3D<float4>(tex[blockIdx.x], 0.1, 0.2, 0.3);
}

