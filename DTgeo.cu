#include "hip/hip_runtime.h"
#include "cuda_math.h"
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <stdlib.h>
#include <unistd.h>
#ifdef MPI_ON
#include <mpi.h>
#endif
#include "omp.h"
#include "params.h"
#include "init.h" 
#include "signal.hpp"
#include "diamond.cu"

__global__ void calc_limits(float* buf, float* fLims) {
  float2 fLim;
  float* pf=buf+blockIdx.x*Nz+threadIdx.x;
  fLim.x = fLim.y = *pf;

  for(int i=0; i<Nx; i++,pf+=Ny*Nz) {
    float v=*pf;
    if(v<fLim.x) fLim.x = v;
    if(v>fLim.y) fLim.y = v;
  }
  __shared__ float2 fLim_sh[Nz];
  fLim_sh[threadIdx.x] = fLim;
  if(threadIdx.x>warpSize) return;
  for(int i=threadIdx.x; i<Nz; i+=warpSize) {
    float2 v=fLim_sh[i];
    if(v.x<fLim.x) fLim.x = v.x;
    if(v.y>fLim.y) fLim.y = v.y;
  }
  fLim_sh[threadIdx.x] = fLim;
  if(threadIdx.x>0) return;
  for(int i=0; i<warpSize; i++) {
    float2 v=fLim_sh[i];
    if(v.x<fLim.x) fLim.x = v.x;
    if(v.y>fLim.y) fLim.y = v.y;
  }
  fLims[2*blockIdx.x  ] = fLim.x;
  fLims[2*blockIdx.x+1] = fLim.y;
}

#include "im2D.h"
#include "im3D.hpp"
int type_diag_flag=0;

im3D_pars im3DHost;

#ifdef USE_TEX_REFS
extern texture<coffS_t, hipTextureType3D, hipReadModeElementType> layerRefS;
extern texture<float  , hipTextureType3D, hipReadModeElementType> layerRefV;
extern texture<float  , hipTextureType3D, hipReadModeElementType> layerRefT;
#endif
texture<coffS_t, hipTextureType3D, hipReadModeElementType> ShowRefS;
texture<float  , hipTextureType3D, hipReadModeElementType> ShowRefV;
texture<float  , hipTextureType3D, hipReadModeElementType> ShowRefT;
void CreateShowTexModel(){
  if(parsHost.texs.ShowTexBinded) return;
  parsHost.texs.ShowTexBinded=1;
  printf("Creating Model texture for im3D showing (size=%.2fMb)\n", parsHost.texs.texN[0].z*parsHost.texs.texN[0].y*parsHost.texs.texN[0].x*sizeof(float)*4/1024./1024. );
  ShowRefS.addressMode[0] = layerRefS.addressMode[0]; ShowRefV.addressMode[0] = layerRefV.addressMode[0]; ShowRefT.addressMode[0] = layerRefT.addressMode[0];
  ShowRefS.addressMode[1] = layerRefS.addressMode[1]; ShowRefV.addressMode[1] = layerRefV.addressMode[1]; ShowRefT.addressMode[1] = layerRefT.addressMode[1];
  ShowRefS.addressMode[2] = layerRefS.addressMode[2]; ShowRefV.addressMode[2] = layerRefV.addressMode[2]; ShowRefT.addressMode[2] = layerRefT.addressMode[2];
  ShowRefS.filterMode = hipFilterModeLinear; ShowRefV.filterMode = layerRefV.filterMode; ShowRefT.filterMode = layerRefT.filterMode;
  ShowRefS.normalized = layerRefS.normalized; ShowRefV.normalized = layerRefV.normalized; ShowRefT.normalized = layerRefT.normalized;

  hipArray* DevModelS, *DevModelV, *DevModelT;
  hipChannelFormatDesc channelDesc;
  channelDesc = hipCreateChannelDesc<coffS_t>(); CHECK_ERROR( hipMalloc3DArray(&DevModelS, &channelDesc, make_hipExtent(parsHost.texs.texN[0].z,parsHost.texs.texN[0].y,parsHost.texs.texN[0].x)) );
  channelDesc = hipCreateChannelDesc<float  >(); CHECK_ERROR( hipMalloc3DArray(&DevModelV, &channelDesc, make_hipExtent(parsHost.texs.texN[0].z,parsHost.texs.texN[0].y,parsHost.texs.texN[0].x)) );
  channelDesc = hipCreateChannelDesc<float  >(); CHECK_ERROR( hipMalloc3DArray(&DevModelT, &channelDesc, make_hipExtent(parsHost.texs.texN[0].z,parsHost.texs.texN[0].y,parsHost.texs.texN[0].x)) );

  const int texNy = parsHost.texs.texN[0].y, texNz = parsHost.texs.texN[0].z;
  hipMemcpy3DParms copyparms={0}; copyparms.srcPos=make_hipPos(0,0,0); copyparms.dstPos=make_hipPos(0,0,0);
  copyparms.kind=hipMemcpyHostToDevice;
  copyparms.srcPtr = make_hipPitchedPtr(&parsHost.texs.HostLayerS[0][0], texNz*sizeof(coffS_t), texNz, texNy);
  copyparms.dstArray = DevModelS;
  copyparms.extent = make_hipExtent(texNz,texNy,parsHost.texs.texN[0].x);
  CHECK_ERROR( hipMemcpy3D(&copyparms) );
  copyparms.srcPtr = make_hipPitchedPtr(&parsHost.texs.HostLayerV[0][0], texNz*sizeof(float  ), texNz, texNy);
  copyparms.dstArray = DevModelV;
  copyparms.extent = make_hipExtent(texNz,texNy,parsHost.texs.texN[0].x);
  CHECK_ERROR( hipMemcpy3D(&copyparms) );
  copyparms.srcPtr = make_hipPitchedPtr(&parsHost.texs.HostLayerT[0][0], texNz*sizeof(float  ), texNz, texNy);
  copyparms.dstArray = DevModelT;
  copyparms.extent = make_hipExtent(texNz,texNy,parsHost.texs.texN[0].x);
  CHECK_ERROR( hipMemcpy3D(&copyparms) );

  channelDesc = hipCreateChannelDesc<coffS_t>(); CHECK_ERROR( hipBindTextureToArray(ShowRefS, DevModelS, channelDesc) );
  channelDesc = hipCreateChannelDesc<float  >(); CHECK_ERROR( hipBindTextureToArray(ShowRefV, DevModelV, channelDesc) );
  channelDesc = hipCreateChannelDesc<float  >(); CHECK_ERROR( hipBindTextureToArray(ShowRefT, DevModelT, channelDesc) );
}
char* FuncStr[] = {"Sx","Sy","Sz","Tx","Ty","Tz", "Vx", "Vy", "Vz", "I1", "I2", "I3", "kappa=l+2m (rho*Vp^2)", "lambda (rho*(Vp^2-2*Vs^2))", 
"mu_Tx (rho*Vs^2)", "mu_Ty (rho*Vs^2)", "mu_Tz (rho*Vs^2)",
"1/rho_Vx", "1/rho_Vy", "1/rho_Vz",
"Index0", "Index1", 
"h_Si", "h_Tx", "h_Ty", "h_Tz", "h_Vx", "h_Vy", "h_Vz" };
__device__ float pow2(float v) { return v*v; }
__device__ double pow2(double v) { return v*v; }
#define MXW_DRAW_ANY(val) *pbuf = val;
__global__ void mxw_draw(float* buf) {
  const ftype d3=1./3; ftype val=0;
  int iz=threadIdx.x;
  DiamondRag* p=&pars.get_plaster(blockIdx.x,blockIdx.y);
  ModelRag* index=&pars.get_index(blockIdx.x,blockIdx.y);
  const int Npls=2*NDT*NDT;
  //#define MODELCOFF_S(text,xv,yv,hv) (tex3D(ShowRefS, (yv)*texStretch[0].y+texShift[0].y, (hv)*texStretchH, int((xv)*texStretchShow.x+texShiftShow.x-0.5)+0.5)*(1-((xv)*texStretchShow.x+texShiftShow.x-0.5-int((xv)*texStretchShow.x+texShiftShow.x-0.5)))+tex3D(ShowRefS, (yv)*texStretch[0].y+texShift[0].y, (hv)*texStretchH, int((xv)*texStretchShow.x+texShiftShow.x-0.5)+1.5)*((xv)*texStretchShow.x+texShiftShow.x-0.5-int((xv)*texStretchShow.x+texShiftShow.x-0.5)))
  //#define MODELCOFF_V(text,xv,yv,hv)  tex3D(ShowRefV, (yv)*texStretch[0].y+texShift[0].y, (hv)*texStretchH, int((xv)*texStretchShow.x+texShiftShow.x-0.5)+0.5)*(1-((xv)*texStretchShow.x+texShiftShow.x-0.5-int((xv)*texStretchShow.x+texShiftShow.x-0.5)))+tex3D(ShowRefV, (yv)*texStretch[0].y+texShift[0].y, (hv)*texStretchH, int((xv)*texStretchShow.x+texShiftShow.x-0.5)+1.5)*((xv)*texStretchShow.x+texShiftShow.x-0.5-int((xv)*texStretchShow.x+texShiftShow.x-0.5))
  //#define MODELCOFF_T(text,xv,yv,hv)  tex3D(ShowRefT, (yv)*texStretch[0].y+texShift[0].y, (hv)*texStretchH, int((xv)*texStretchShow.x+texShiftShow.x-0.5)+0.5)*(1-((xv)*texStretchShow.x+texShiftShow.x-0.5-int((xv)*texStretchShow.x+texShiftShow.x-0.5)))+tex3D(ShowRefT, (yv)*texStretch[0].y+texShift[0].y, (hv)*texStretchH, int((xv)*texStretchShow.x+texShiftShow.x-0.5)+1.5)*((xv)*texStretchShow.x+texShiftShow.x-0.5-int((xv)*texStretchShow.x+texShiftShow.x-0.5))
  #ifndef CUDA_TEX_INTERP
  #else
  #define DISTS(xt,yt,zt) sqrtf((xt-CntX)*(xt-CntX)+(yt-CntY)*(yt-CntY)+(zt-CntZ)*(zt-CntZ))
  #define MODELCOFF_S(text,xv,yv,zv) tex3D(text, DISTS(xv,yv,zv)*texStretch[0].z+texShift[0].z, 0,0)
  #endif
  #define MODELCOFF_V(text,xv,yv,hv) MODELCOFF_S(text,xv,yv,hv) 
  #define MODELCOFF_T(text,xv,yv,hv) MODELCOFF_S(text,xv,yv,hv) 
  for(int idom=0; idom<Npls; idom++) {
    int Ragdir=0;
    if(pars.nFunc==6 || pars.nFunc==7 || pars.nFunc==8 || pars.nFunc==17 || pars.nFunc==18 || pars.nFunc==19 || pars.nFunc==26 || pars.nFunc==27 || pars.nFunc==28 ) Ragdir=1;
    int shx=-NDT+idom/NDT+idom%NDT;
    int shy=+NDT-idom/NDT+idom%NDT;
    if(Ragdir) shy=0+idom/NDT-idom%NDT;
    int idev=0, nextY=NStripe(0);
    #if 1
      while(blockIdx.y>=nextY) nextY+=NStripe(++idev);
      shy-=idev*2*NDT;
      if(blockIdx.y==nextY-NStripe(idev) && idom< Npls/2 && Ragdir==1 && idev!=0     ) continue;
      if(blockIdx.y==nextY-NStripe(idev) && idom>=Npls/2 && Ragdir==0 && idev!=0     ) continue;
      if(blockIdx.y==nextY-1             && idom< Npls/2 && Ragdir==0 && idev!=NDev-1) continue;
      if(blockIdx.y==nextY-1             && idom>=Npls/2 && Ragdir==1 && idev!=NDev-1) continue;
    #endif
    int ix = blockIdx.x*2*NDT+shx+4;
    int iy = blockIdx.y*2*NDT+shy+2;
    float* pbuf=&buf[threadIdx.x+NT*((iy/2)%(Ny-1)+Ny*(ix/2))];
    switch(pars.nFunc) {
      case 0 : if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(p->Si[idom/2].duofld[0][iz].x ); } break; //Sx
      case 1 : if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(p->Si[idom/2].duofld[0][iz].y ); } break; //Sy
      case 2 : if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(p->Si[idom/2].duofld[1][iz].x ); } break; //Sz
      case 4 : if(idom%2==0) { pbuf+=NT*Ny; MXW_DRAW_ANY(p->Si[idom/2].duofld[1][iz].y ); } break; //Ty
      case 5 : if(idom%2==1) { pbuf+=0    ; MXW_DRAW_ANY(p->Si[idom/2].duofld[2][iz].x ); } break; //Tz
      case 3 : if(idom%2==1) { pbuf+=0    ; MXW_DRAW_ANY(p->Si[idom/2].duofld[2][iz].y ); } break; //Tx
      case 7 : if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(p->Vi[idom/2].trifld.one[iz]  ); } break; //Vy
      case 6 : if(idom%2==1) { pbuf+=0    ; MXW_DRAW_ANY(p->Vi[idom/2].trifld.two[iz].x); } break; //Vx
      case 8 : if(idom%2==1) { pbuf+=0    ; MXW_DRAW_ANY(p->Vi[idom/2].trifld.two[iz].y); } break; //Vz
      case 9 : if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(d3*(p->Si[idom/2].duofld[0][iz].x+p->Si[idom/2].duofld[0][iz].y+p->Si[idom/2].duofld[1][iz].x)); } break; //Inv1
      case 10: if(idom%2==0) { pbuf+=0    ; ftype Sx = p->Si[idom/2].duofld[0][iz].x; ftype Sy = p->Si[idom/2].duofld[0][iz].y; ftype Sz = p->Si[idom/2].duofld[1][iz].x; atomicAdd(pbuf, Sx*Sy+Sx*Sz+Sy*Sz); }
               if(idom%2==1) { pbuf+=0    ; ftype Tx = p->Si[idom/2].duofld[2][iz].y; atomicAdd(pbuf,-0.25*Tx*Tx);if(iz  >0)atomicAdd(pbuf-1 ,-0.25*Tx*Tx);if(iy/2>0)atomicAdd(pbuf-NT   ,-0.25*Tx*Tx);if(iz  >0 && iy/2>0)atomicAdd(pbuf-NT   -1 ,-0.25*Tx*Tx); } 
               if(idom%2==0) { pbuf+=NT*Ny; ftype Ty = p->Si[idom/2].duofld[1][iz].y; atomicAdd(pbuf,-0.25*Ty*Ty);if(iz  >0)atomicAdd(pbuf-1 ,-0.25*Ty*Ty);if(ix/2>0)atomicAdd(pbuf-NT*Ny,-0.25*Ty*Ty);if(iz  >0 && ix/2>0)atomicAdd(pbuf-NT*Ny-1 ,-0.25*Ty*Ty); } 
               if(idom%2==1) { pbuf+=0    ; ftype Tz = p->Si[idom/2].duofld[2][iz].x; atomicAdd(pbuf,-0.25*Tz*Tz);if(iy/2>0)atomicAdd(pbuf-NT,-0.25*Tz*Tz);if(ix/2>0)atomicAdd(pbuf-NT*Ny,-0.25*Tz*Tz);if(iy/2>0 && ix/2>0)atomicAdd(pbuf-NT*Ny-NT,-0.25*Tz*Tz); } break; //inv2
      case 11: break; //inv3
      case 12: if(idom%2==0)              { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_S(ShowRefS, ix, iy, (iz*2+1)).x); } break; // Vp*Vp*rho
      case 13: if(idom%2==0)              { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_S(ShowRefS, ix, iy, (iz*2+1)).y); } break; // (Vp*Vp-2*Vs*Vs)*rho
      case 14: if(idom%2==1)              { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_T(ShowRefT, ix, iy, (iz*2+1))  ); } break; // Vs*Vs*rho for T_x
      case 15: if(idom%2==0)              { pbuf+=NT*Ny; MXW_DRAW_ANY(MODELCOFF_T(ShowRefT, ix, iy, (iz*2+1))  ); } break; // Vs*Vs*rho for T_y
      case 16: if(idom%2==1)              { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_T(ShowRefT, ix, iy, (iz*2+1))  ); } break; // Vs*Vs*rho for T_z
      case 17: if(idom%2==1 && (idom+1)%4==0) { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_V(ShowRefV, ix, iy, (iz*2+1))); } 
          else if(idom%2==1 && (idom+1)%4!=0) { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_V(ShowRefV, ix, iy, (iz*2+1))); } break; // 1/rho for V_x
      case 18: if(idom%2==0 && idom    %4==0) { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_V(ShowRefV, ix, iy, (iz*2+1))); } 
          else if(idom%2==0 && idom    %4!=0) { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_V(ShowRefV, ix, iy, (iz*2+1))); } break; // 1/rho for V_y
      case 19: if(idom%2==1 && (idom+1)%4!=0) { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_V(ShowRefV, ix, iy, (iz*2+1))); } 
          else if(idom%2==1 && (idom+1)%4==0) { pbuf+=0    ; MXW_DRAW_ANY(MODELCOFF_V(ShowRefV, ix, iy, (iz*2+1))); } break; // 1/rho for V_z
//      case 20: if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(index->sInd[(idom<NDT*NDT)?0:1][iz]); } break; //Index0
//      case 21: if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(index->sInd[(idom<NDT*NDT)?2:3][iz]); } break; //Index1
//      case 22: if(idom%2==0) { pbuf+=0    ; MXW_DRAW_ANY(*((float*)&(index->h[idom/2*2  ][iz].x))); } break; //h_Si
//      case 24: if(idom%2==0) { pbuf+=NT*Ny; MXW_DRAW_ANY(float(*((double*)&(index->h[idom/2*2  ][iz].y)))); } break; //h_Ty
//      case 25: if(idom%2==1) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[idom/2*2+1][iz].x)))); } break; //h_Tz
//      case 23: if(idom%2==1) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[idom/2*2+1][iz].y)))); } break; //h_Tx
//      case 27: if(idom%2==0 && idom%4==0    ) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[Npls+idom/2*3/2    ][iz].x)))); } 
//          else if(idom%2==0 && idom%4!=0    ) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[Npls+idom/2*3/2    ][iz].y)))); }; break; //h_Vy
//      case 26: if(idom%2==1 && (idom+1)%4==0) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[Npls+(idom/2*3+1)/2][iz].x)))); }
//          else if(idom%2==1 && (idom+1)%4!=0) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[Npls+(idom/2*3+1)/2][iz].y)))); }; break; //h_Vx
//      case 28: if(idom%2==1 && (idom+1)%4!=0) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[Npls+(idom/2*3+2)/2][iz].x)))); }
//          else if(idom%2==1 && (idom+1)%4==0) { pbuf+=0    ; MXW_DRAW_ANY(float(*((double*)&(index->h[Npls+(idom/2*3+2)/2][iz].y)))); }; break; //h_Vz
    }
    if(pars.nFunc<12 && pars.bgMat) if(idom%2==0) { pbuf+=0    ; atomicAdd(pbuf, 5*pow(0.1,double(7-pars.bgMat))*MODELCOFF_S(ShowRefS, ix, iy, (iz*2+1)).y); }
  }
  switch(pars.nFunc) {
//    case 10: val=1./3*(c->Sx[iz]*c->Sy[iz]+c->Sx[iz]*c->Sz[iz]+c->Sy[iz]*c->Sz[iz]-c->Tx[iz]*c->Tx[iz]-c->Ty[iz]*c->Ty[iz]-c->Tz[iz]*c->Tz[iz]); MXW_DRAW_ANY(val>0?sqrt(val):-sqrt(-val)); break;
//    case 11: val=1./3*(c->Sx[iz]*c->Sy[iz]*c->Sz[iz]+2*c->Tx[iz]*c->Ty[iz]*c->Tz[iz]-c->Sx[iz]*c->Tx[iz]*c->Tx[iz]-c->Sy[iz]*c->Ty[iz]*c->Ty[iz]-c->Sz[iz]*c->Tz[iz]*c->Tz[iz]); MXW_DRAW_ANY(pow(val, 1.f/3.f)); break;
//    case 12: MXW_DRAW_ANY(TEXVp (blockIdx.x,blockIdx.y,threadIdx.x )); break;
//    case 13: MXW_DRAW_ANY(TEXVs (blockIdx.x,blockIdx.y,threadIdx.x )); break;
//    case 14: MXW_DRAW_ANY(TEXRho(blockIdx.x,blockIdx.y,threadIdx.x )); break;
    default: break;
  }
}

struct any_idle_func_struct {
    virtual void step() {}
};
struct idle_func_calc: public any_idle_func_struct {
  float t;
  void step();
};
void idle_func_calc::step() {
  calcStep();
  CreateShowTexModel();
  CHECK_ERROR( hipMemset(parsHost.arr4im.Arr3Dbuf,0,((long long int)Nx)*Ny*Nz*sizeof(ftype)) );
  mxw_draw<<<dim3((USE_UVM==2)?Np:Ns,Na),NT>>>(parsHost.arr4im.Arr3Dbuf);
  im3DHost.initCuda(parsHost.arr4im);
  recalc_at_once=true;
}

unsigned char* read_config_file(int& n){
  n = 0; int c; 
  FILE* cfgfile;
  cfgfile = fopen("acts.cfg","r");
  if (cfgfile==NULL) return NULL;
  else {
    c = fgetc(cfgfile); if(c == EOF) {printf("config file is empty"); return NULL; } 
    n = 0;
    while(c != EOF) {
      c = fgetc(cfgfile);
      n++;
    }
    fclose(cfgfile);
  }
  unsigned char* actlist = NULL;
  cfgfile = fopen("acts.cfg","r");
  if (cfgfile==NULL) return NULL;
  else {
    actlist = new unsigned char[n];
    for(int i=0; i<n; i++) { 
      actlist[i] = (unsigned char)fgetc(cfgfile);
      if     (actlist[i]=='\n') actlist[i] = 13;
      else if(actlist[i]=='2' ) actlist[i] = 50;
      else if(actlist[i]=='3' ) actlist[i] = 51;
    }
    fclose(cfgfile);
  }
  return actlist; 
}
int iact = 0;
int nact = 0;
unsigned char* sequence_act = NULL; 
static void key_func(unsigned char key, int x, int y) {
  if(type_diag_flag>=2) printf("keyN=%d, coors=(%d,%d)\n", key, x, y);
  if(key == 'h') {
    printf("\
======= Управление mxw3D:\n\
  <¦>  \tИзменение функции для визуализации: WEH¦Sx¦Ez¦Ey¦Ex¦Hx¦Hy¦Hz¦Sy¦Sz¦eps\n\
«Enter»\tПересчёт одного большого шага\n\
   b   \tвключает пересчёт в динамике (см. «Управление динамикой»)\n\
"); im3DHost.print_help();
    return;
  }
  switch(key) {
  //case '>': if(parsHost.nFunc<parsHost.MaxFunc) parsHost.nFunc++; break;
  //case '<': if(parsHost.nFunc>0) parsHost.nFunc--; break;
  case '>': parsHost.nFunc = (parsHost.nFunc+1)%parsHost.MaxFunc; break;
  case '<': parsHost.nFunc = (parsHost.nFunc+parsHost.MaxFunc-1)%parsHost.MaxFunc; break;
  case 'B': parsHost.bgMat = (parsHost.bgMat+1)%7; break;
  case 13: calcStep(); break;
  //case  8: recalc_at_once=arr3D_list->prev_arr2gpu(); return;
  case 'c': 
    {
    printf("reading config file\n");
    sequence_act = read_config_file(nact);
    glutPostRedisplay();
    return; 
    }
  default: if(!im3DHost.key_func(key, x, y)) {
  if(type_diag_flag>=0) printf("По клавише %d в позиции (%d,%d) нет никакого действия\n", key, x, y);
  } return;
  }
  copy2dev( parsHost, pars );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  CreateShowTexModel();
  CHECK_ERROR( hipMemset(parsHost.arr4im.Arr3Dbuf,0,((long long int)Nx)*Ny*Nz*sizeof(ftype)) );
  mxw_draw<<<dim3((USE_UVM==2)?Np:Ns,Na),NT>>>(parsHost.arr4im.Arr3Dbuf);
  im3DHost.initCuda(parsHost.arr4im);
  recalc_at_once=true;
}
static void draw_func() { 
  if (iact<nact) { 
    key_func(sequence_act[iact],0,0);
    iact++;
    glutPostRedisplay();
  }
  if(nact>0 && iact==nact) delete[] sequence_act;
  im3DHost.fName = FuncStr[parsHost.nFunc]; im2D.draw(im3DHost.reset_title()); 
}

//void (*idle_func_ptr)(float* );
static void idle_func() { im3DHost.recalc_func(); }
static void mouse_func(int button, int state, int x, int y) { im3DHost.mouse_func(button, state, x, y); }
static void motion_func(int x, int y) { im3DHost.motion_func(x, y); }

double PMLgamma_func(int i, int N, ftype dstep){ //return 0; 
  if(i>=N-3) return 0;
  N-=3;
  double attenuation_factor = 4;
  double sigma_max= shotpoint.V_max*log(1000)*( (attenuation_factor+1)/(2*(N*dstep*0.5)) );
  double x_max = pow(sigma_max, 1./attenuation_factor);
  double x = x_max-i*(x_max/N);
  return pow(x, attenuation_factor);
}
double PMLgamma_funcY(int i, int N, ftype dstep){ //return 0;
  if(i>=N-3) return 0;
  N-=3;
  double attenuation_factor = 4;
  double sigma_max= shotpoint.V_max*log(1000)*( (attenuation_factor+1)/(2*(N*dstep*0.5)) );
  double x_max = pow(sigma_max, 1./attenuation_factor);
  double x = x_max-i*(x_max/N);
  return pow(x, attenuation_factor);
}
double PMLgamma_funcZ(int i, int N, ftype dstep){ //return 0; 
  if(i>=N-3) return 0;
  N-=3;
  double attenuation_factor = 4;
  double sigma_max= shotpoint.V_max*log(1000)*( (attenuation_factor+1)/(2*(N*dstep*0.5)) );
  double x_max = pow(sigma_max, 1./attenuation_factor);
  double x = x_max-i*(x_max/N);
  return pow(x, attenuation_factor);
}
void setPMLcoeffs(ftype* k1x, ftype* k2x, ftype* k1y, ftype* k2y, ftype* k1z, ftype* k2z) {
  for(int i=0; i<KNpmlx; i++){
    k2x[i] = 1.0/(1.0+0.5*dt*PMLgamma_func(KNpmlx/2-abs(i-KNpmlx/2)-3, KNpmlx/2-3, dx));
    k1x[i] = 2.0*k2x[i]-1;
  }
  for(int i=0; i<KNpmly; i++){
    //k2y[i] = 1.0/(1.0+0.5*dt*PMLgamma_funcY(KNpmly-i, KNpmly, dy));
    k2y[i] = 1.0/(1.0+0.5*dt*PMLgamma_func(KNpmly/2-abs(i-KNpmly/2), KNpmly/2, dy));
    k1y[i] = 2.0*k2y[i]-1;
  }
  for(int i=0; i<KNpmlz; i++){
    k2z[i] = 1.0/(1.0+0.5*dt*PMLgamma_funcZ(KNpmlz/2-abs(i-KNpmlz/2), KNpmlz/2, dz));
    k1z[i] = 2.0*k2z[i]-1;
  }
}
void setPeer2Peer(int node,int subnode, int* isp2p){
  for(int i=0; i<NDev; i++) for(int j=i+1; j<NDev; j++) {
      int canp2p=0; CHECK_ERROR(hipDeviceCanAccessPeer(&canp2p,i,j));
      if(canp2p) { 
        CHECK_ERROR(hipSetDevice(i)); CHECK_ERROR(hipDeviceEnablePeerAccess(j,0));
        CHECK_ERROR(hipSetDevice(j)); CHECK_ERROR(hipDeviceEnablePeerAccess(i,0));
              printf("node.subnode %d.%d: %d<-->%d can Peer2Peer\n"   , node, subnode, i,j);
      } else  printf("node.subnode %d.%d: %d<-->%d cannot Peer2Peer\n", node, subnode, i,j);
      if(j==i+1) isp2p[i]=canp2p;
  }
  CHECK_ERROR(hipSetDevice(0)); 
}
void GeoParamsHost::set(){
  
  #ifndef USE_WINDOW
  if(Np!=Ns) { printf("Error: if not defined USE_WINDOW Np must be equal Ns\n"); exit(-1); }
  #endif//USE_WINDOW

  node=0; subnode=0; int Nprocs=1;
  #ifdef MPI_ON
  MPI_Comm_rank (MPI_COMM_WORLD, &node);   subnode=node%NasyncNodes; node/= NasyncNodes;
  MPI_Comm_size (MPI_COMM_WORLD, &Nprocs); 
  if(node==0) printf("Total MPI tasks: %d\n", Nprocs);
  #endif
  if(Nprocs%NasyncNodes!=0) { printf("Error: mpi procs (%d) must be dividable by NasyncNodes(%d)\n",Nprocs,NasyncNodes); exit(-1); }
  Nprocs/= NasyncNodes;
  mapNodeSize = new int[Nprocs];
  int accSizes=0;
  mapNodeSize[0] = Np/Nprocs+Ns/2; for(int i=1; i<Nprocs; i++) mapNodeSize[i] = Np/Nprocs+Ns;
  int sums=0; for(int i=0; i<Nprocs-1; i++) sums+= mapNodeSize[i]-Ns; mapNodeSize[Nprocs-1]=Np-sums;
  for(int i=0; i<Nprocs; i++) {
    if(node==i) printf("X-size=%d rags on node %d\n", mapNodeSize[i], i);
    #ifdef MPI_ON
    if(mapNodeSize[i]<2*Ns && Nprocs>1) { printf("Data on node %d is too small\n", i); exit(-1); }
    #endif
    accSizes+= mapNodeSize[i];
  }
  if(accSizes-Ns*(Nprocs-1)!=Np) { printf("Error: sum (mapNodes) must be = Np+Ns*(Nprocs-1)\n"); exit(-1); }
  #ifdef MPI_ON
  if(mapNodeSize[0]       <=Npmlx/2+Ns+Ns && Nprocs>1) { printf("Error: mapNodeSize[0]<=Npmlx/2+Ns+Ns\n"); exit(-1); }
  if(mapNodeSize[Nprocs-1]<=Npmlx/2+Ns+Ns && Nprocs>1) { printf("Error: mapNodeSize[Nodes-1]<=Npmlx/2+Ns+Ns\n"); exit(-1); }
  #endif
  if(Np%Ns!=0) { printf("Error: Np must be dividable by Ns\n"); exit(-1); }
  if(NB%NA!=0) { printf("Error: NB must be dividable by NA\n"); exit(-1); }
  if(NB<NA   ) { printf("Error: NB < NA\n"); exit(-1); }
  omp_set_num_threads(8);

  //dir= new string("/Run/zakirov/tmp/"); //ix=Nx+Nbase/2; Yshtype=0;
  dir= new std::string(im3DHost.drop_dir);
  drop.dir=dir;
  struct stat st = {0};

  if (stat(dir->c_str()     , &st) == -1)  mkdir(dir->c_str()     , 0700);
  if (stat(swap_dir->c_str(), &st) == -1)  mkdir(swap_dir->c_str(), 0700);
  
  for(int i=0;i<NDev-1;i++) isp2p[i]=0;
  setPeer2Peer(node,subnode,isp2p);
  for(int i=0;i<NDev-1;i++) isp2p[i]=1;
  
  if(node==0) print_info();
  Tsteps+= node*Ntime;
  if(node==0) printf("Full %d Big steps\n", Tsteps/Ntime);
  if(node==0) printf("Grid size: %dx%d Rags /%dx%dx%d Yee_cells/, TorreH=%d\n", Np, Na, Np*NDT,Na*NDT,Nv, Ntime);
  if(node==0) printf("Window size: %d, copy-shift step %d \n", Ns, Window::NTorres );
  if(gridNx%NDT!=0) { printf("Error: gridNx must be dividable by %d\n", NDT); exit(-1); }
  if(gridNy%NDT!=0) { printf("Error: gridNy must be dividable by %d\n", NDT); exit(-1); }
  if(dt*sqrt(1/(dx*dx)+1/(dy*dy)+1/(dz*dz))>6./7.) { printf("Error: Courant condition is not satisfied\n"); exit(-1); }
//  if(sizeof(DiamondRag)!=sizeof(RagArray)) { printf("Error: sizeof(DiamondRag)=%d != sizeof(RagArray)\n", sizeof(DiamondRag),sizeof(RagArray)); exit(-1); }
  int NaStripe=0; for(int i=0;i<NDev;i++) NaStripe+=NStripe[i]; if(NaStripe!=Na) { printf("Error: sum(NStripes[i])!=NA\n"); exit(-1); }
  iStep = 0; isTFSF=true;
  Zcnt=0.5*Nz*dz;
  //IndNx=2*(Np*NDT+2); IndNy=2*(Na*NDT+1); IndNz=2*Nz;
  #ifdef COFFS_DEFAULT
  //IndNx=1; IndNy=1; IndNz=1;
  #endif
  nFunc = 0; MaxFunc = sizeof(FuncStr)/sizeof(char*);
  size_t size_xz     = Ns   *sizeof(DiamondRag   );
  size_t size_xzModel= Ns   *sizeof(ModelRag     );
  size_t sz          = Na*size_xz;
  size_t szModel     = Na*size_xzModel;
  size_t szBuf       = Ntime*sizeof(halfRag      );
  size_t szPMLa      = Ns*Npmly*sizeof(DiamondRagPML);
  size_t size_xzPMLs = Npmlx/2*sizeof(DiamondRagPML);
  #ifdef NOPMLS
  size_xzPMLs = 0;
  #endif
  size_t szPMLs      = Na*size_xzPMLs;
  if(node==0) {
  printf("GPU Cell's Array size     : %7.2fM = %7.2fM(Main)+%7.2fM(Buffers)+%7.2fM(Model)+%7.2fM(PMLs)+%7.2fM(PMLa)\n", 
           (sz+2*NDev*szBuf+szModel+szPMLs+szPMLa)/(1024.*1024.),
           sz     /(1024.*1024.), 
      NDev*szBuf*2/(1024.*1024.), 
           szModel/(1024.*1024.), 
           szPMLs /(1024.*1024.), 
           szPMLa /(1024.*1024.)  );
  for(int istrp=0; istrp<NDev-1; istrp++) printf( "                   Stripe%d: %7.2fM = %7.2fM      +%7.2fM         +%7.2fM       +%7.2fM\n", istrp, 
           (size_xz*NStripe[istrp]+2*szBuf+size_xzModel*NStripe[istrp]+size_xzPMLs*NStripe[istrp])/(1024.*1024.),
           size_xz    *NStripe[istrp ]/(1024.*1024.), 
           szBuf*2                    /(1024.*1024.), 
           size_xzModel*NStripe[istrp]/(1024.*1024.),
           size_xzPMLs*NStripe[istrp ]/(1024.*1024.)  );
                                          printf( "                   Stripe%d: %7.2fM = %7.2fM      +%7.2fM         +%7.2fM       +%7.2fM      +%7.2fM\n", NDev-1, 
           (size_xz*NStripe[NDev-1]+2*szBuf+size_xzModel*NStripe[NDev-1]+size_xzPMLs*NStripe[NDev-1]+szPMLa)/(1024.*1024.),
           size_xz    *NStripe[NDev-1]/(1024.*1024.), 
           szBuf*2                    /(1024.*1024.), 
           size_xzModel*NStripe[NDev-1]/(1024.*1024.),
           size_xzPMLs*NStripe[NDev-1]/(1024.*1024.), 
           szPMLa                      /(1024.*1024.)  );
  }
  fflush(stdout);
  size_t freemem[NDev], totalmem[NDev];
  for(int idev=0; idev<NDev; idev++) {
    CHECK_ERROR( hipSetDevice(idev) );
    CHECK_ERROR( hipMalloc( (void**)&(ragsInd  [idev]), size_xzModel*NStripe[idev]) );
    CHECK_ERROR( hipMalloc( (void**)&(rags     [idev]), size_xz     *NStripe[idev]) );
    CHECK_ERROR( hipMalloc( (void**)&(p2pBufM  [idev]), szBuf     ) );
    CHECK_ERROR( hipMalloc( (void**)&(p2pBufP  [idev]), szBuf     ) );
//    CHECK_ERROR( hipMalloc( (void**)&(ragsPMLs[idev]), size_xzPMLs*NStripe[idev]    ) );
    #ifndef USE_WINDOW
    CHECK_ERROR( hipMalloc( (void**)&(ragsPMLsL[idev]), size_xzPMLs*NStripe[idev]    ) );
    CHECK_ERROR( hipMalloc( (void**)&(ragsPMLsR[idev]), size_xzPMLs*NStripe[idev]    ) );
    #endif
    CHECK_ERROR( hipMalloc( (void**)& ragsPMLa[idev]  , szPMLa ) );
    CHECK_ERROR( hipMemset(rags    [idev], 0, size_xz    *NStripe[idev])  );
    CHECK_ERROR( hipMemset(p2pBufM [idev], 0, szBuf)  );
    CHECK_ERROR( hipMemset(p2pBufP [idev], 0, szBuf)  );
    CHECK_ERROR( hipMemset(ragsInd [idev], 0, size_xzModel*NStripe[idev]) );
    #ifndef USE_WINDOW
    hipMemset(ragsPMLsL[idev], 0,  size_xzPMLs*NStripe[idev]);
    hipMemset(ragsPMLsR[idev], 0,  size_xzPMLs*NStripe[idev]);
    #endif
    if(idev==NDev-1)
    CHECK_ERROR( hipMemset(ragsPMLa[idev], 0,                     szPMLa) );
    CHECK_ERROR( hipMemGetInfo(&freemem[idev], &totalmem[idev]));
    printf("Node/subnode %3d/%d : device %d: GPU memory free %.2fM of %.2fM\n", node, subnode, idev, freemem[idev]/(1024.*1024.), totalmem[idev]/(1024.*1024.) );
  }
  CHECK_ERROR( hipSetDevice(0) );
  fflush(stdout);

  const int Nn = mapNodeSize[node];
  #if 1//USE_WINDOW
  printf("Allocating RAM memory on node %d: %g Gb\n", node, (Nn*Na*sizeof(DiamondRag)+Nn*Na*sizeof(ModelRag)+Nn*Npmly*sizeof(DiamondRagPML)+Npmlx*Na*sizeof(DiamondRagPML))/(1024.*1024.*1024.));
  fflush(stdout);
  #if USE_UVM==2
  #ifdef SWAP_DATA
  char swapdata[256]; sprintf(swapdata, "%s/swapdata.%d.%d", swap_dir->c_str(), node,subnode);
  int swp_data; swp_data = open(swapdata,O_RDWR|O_TRUNC|O_CREAT, 0666);
  if(swp_data==-1) { char s[128]; sprintf(s,"Error opening file %s at %d.%d",swapdata,node,subnode); perror(s); exit(-1); }
  lseek(swp_data, Nn*Na*sizeof(DiamondRag), SEEK_SET);
  write(swp_data, "", 1); lseek(swp_data, 0, SEEK_SET);
  data = (DiamondRag*)mmap(0, Nn*Na*sizeof(DiamondRag), PROT_READ|PROT_WRITE, MAP_SHARED, swp_data,0);
  if(data == MAP_FAILED) { char s[128]; sprintf(s,"Error mmap data at %d.%d",node,subnode); perror(s); exit(-1); }
  close(swp_data);
  #else
  CHECK_ERROR( hipHostMalloc(&data     , Nn*Na     *sizeof(DiamondRag    )) );
  #endif//SWAP_DATA
  memset(data     , 0, Nn*Na     *sizeof(DiamondRag    ));
  CHECK_ERROR( hipHostMalloc(&dataInd  , Nn*Na     *sizeof(ModelRag      )) ); memset(dataInd  , 0, Nn*Na     *sizeof(ModelRag      ));
  CHECK_ERROR( hipHostMalloc(&dataPMLa , Nn*Npmly  *sizeof(DiamondRagPML )) ); memset(dataPMLa , 0, Nn*Npmly  *sizeof(DiamondRagPML ));
  CHECK_ERROR( hipHostMalloc(&dataPMLsL, Npmlx/2*Na*sizeof(DiamondRagPML )) ); memset(dataPMLsL, 0, Npmlx/2*Na*sizeof(DiamondRagPML ));
  CHECK_ERROR( hipHostMalloc(&dataPMLsR, Npmlx/2*Na*sizeof(DiamondRagPML )) ); memset(dataPMLsR, 0, Npmlx/2*Na*sizeof(DiamondRagPML ));
  if (node==1) printf("data allocated, pointer to %p\n", data);
  for(int i=0; i<node; i++) data    -= mapNodeSize[i]*Na   ; data    +=node*Ns*Na;
  for(int i=0; i<node; i++) dataInd -= mapNodeSize[i]*Na   ; dataInd +=node*Ns*Na;
  for(int i=0; i<node; i++) dataPMLa-= mapNodeSize[i]*Npmly; dataPMLa+=node*Ns*Npmly;
  if (node==1) printf("now data points to %p\n", data);
  fflush(stdout);
  #else
  data     = new DiamondRag   [Nn*Na   ]; memset(data    , 0, Nn*Na   *sizeof(DiamondRag   ));
  dataPMLa = new DiamondRagPML[Nn*Npmly]; memset(dataPMLa, 0, Nn*Npmly*sizeof(DiamondRagPML));
  dataPMLs = new DiamondRagPML[Npmlx*Na]; memset(dataPMLs, 0, Npmlx*Na*sizeof(DiamondRagPML));
  #endif
  #endif
  //size_t size_rdma = sizeof(DiamondRag)*(NDT*NDT/2+1);
  size_t size_rdma = szBuf;
  CHECK_ERROR( hipHostMalloc( (void**)&rdma_send_buf, size_rdma ) );
  CHECK_ERROR( hipHostMalloc( (void**)&rdma_recv_buf, size_rdma ) );
  for(int idev=0; idev<NDev; idev++) {
    CHECK_ERROR( hipHostMalloc( (void**)&(p2pBufM_host_snd[idev]), szBuf     ) );
    CHECK_ERROR( hipHostMalloc( (void**)&(p2pBufP_host_snd[idev]), szBuf     ) );
    CHECK_ERROR( hipHostMalloc( (void**)&(p2pBufM_host_rcv[idev]), szBuf     ) );
    CHECK_ERROR( hipHostMalloc( (void**)&(p2pBufP_host_rcv[idev]), szBuf     ) );
    CHECK_ERROR( hipMemset(p2pBufM_host_snd[idev], 0, szBuf)  );
    CHECK_ERROR( hipMemset(p2pBufP_host_snd[idev], 0, szBuf)  );
    CHECK_ERROR( hipMemset(p2pBufM_host_rcv[idev], 0, szBuf)  );
    CHECK_ERROR( hipMemset(p2pBufP_host_rcv[idev], 0, szBuf)  );
  }
  for(int i=0; i<NDev-1; i++) {
    size_t size_p2p = sizeof(DiamondRag)*(NDT*NDT/2+1);
    p2p_buf[i]=0;
    if(isp2p[i]) CHECK_ERROR( hipHostMalloc( (void**)&p2p_buf[i], size_p2p ) );
  }

  drop.init();
  texs.init();
  cuTimer t0; t0.init();
  int xL=0; for(int inode=0; inode<node; inode++) xL+= mapNodeSize[inode]; xL-= Ns*node;
  int xR = xL+mapNodeSize[node];
  omp_set_num_threads(8);
  for(int x=0;x<Np;x++) {
    printf("Initializing h-parameter %.2f%%      \r",100*double(x+1)/Np); fflush(stdout);
    if(x>=xL && x<xR) { 
      #pragma omp parallel for
      for(int y=0;y<Na;y++) {
        //printf("Initializing h-parameter %.2f%%      \r",100*double(x*Na+y+1)/Np/Na); fflush(stdout);
        dataInd[x*Na+y].set(x,y);
      }
    }
  }
//  printf("t0=%g\n",t0.gettime());
  
  sensors = new std::vector<Sensor>();
}

hipArray* radTexS_array, *radTexV_array, *radTexT_array;

void init_index() {
  //-------Set PML coeffs----------------------------//
  hostKpmlx1 = new ftype[KNpmlx]; hostKpmlx2 = new ftype[KNpmlx];
  hostKpmly1 = new ftype[KNpmly]; hostKpmly2 = new ftype[KNpmly];
  hostKpmlz1 = new ftype[KNpmlz]; hostKpmlz2 = new ftype[KNpmlz];
  setPMLcoeffs(hostKpmlx1, hostKpmlx2, hostKpmly1, hostKpmly2, hostKpmlz1, hostKpmlz2);
  for(int idev=0; idev<NDev; idev++) {
    CHECK_ERROR( hipSetDevice(idev) );
    CHECK_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(Kpmlx1), hostKpmlx1, sizeof(ftype)*KNpmlx) );
    CHECK_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(Kpmlx2), hostKpmlx2, sizeof(ftype)*KNpmlx) );
    CHECK_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(Kpmly1), hostKpmly1, sizeof(ftype)*KNpmly) );
    CHECK_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(Kpmly2), hostKpmly2, sizeof(ftype)*KNpmly) );
    CHECK_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(Kpmlz1), hostKpmlz1, sizeof(ftype)*KNpmlz) );
    CHECK_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(Kpmlz2), hostKpmlz2, sizeof(ftype)*KNpmlz) );
  }
  CHECK_ERROR( hipSetDevice(0) );
  //-----------------------------------------------------------------------------------//

/*  parsHost.sensors->push_back(Sensor("Ez",(X0+Rreson)/dx+1,Y0/dy,Z0/dz));
  parsHost.sensors->push_back(Sensor("Ez",(X0-Rreson)/dx-1,Y0/dy,Z0/dz));
  parsHost.sensors->push_back(Sensor("Ez",X0/dx,(Y0+Rreson)/dy+1,Z0/dz));
  parsHost.sensors->push_back(Sensor("Ez",X0/dx,(Y0-Rreson)/dy-1,Z0/dz));*/
}
void set_texture(const int ix=0){
}

int print_help() {
  printf("using: ./DFmxw [--help] [--zoom \"1. 1. 1.\"] [--step \"1. 1. 1.\"] [--box \"1. 1. 1.\"] [--mesh \"200. 200. 200.\"] [--Dmesh 5.] [--drop_dir \".\"] [--bkgr_col \"0.1 0.1 0.1\"] [--mesh_col \"0.8 0.8 0.2\"] [--box_col \"1. 1. 1.\"] [--sensor \"1 1 1\"]\n");
  printf("  --zoom\tмасштабный фактор, действует на 2D режим и размер окна, [1. 1. 1.];\n");
  printf("  --box \tкоррекция пропорций размера бокса в 3D режиме, [1. 1. 1.];\n");
  printf("  --step \tшаги между точками, действует только на тики, [1. 1. 1.];\n");
  printf("  --mesh\tрасстояние между линиями сетки в боксе по координатам в ячейках (до коррекции), [200. 200. 200.];\n");
  printf("  --Dmesh\tширина линии сетки в пикселях (со сглаживанием выглядит несколько уже), [5.];\n");
  printf("  --drop_dir\tимя директории, в которую будут сохраняться различные файлы, [.];\n");
  printf("  --bkgr_col\tцвет фона, [0.1 0.1 0.1];\n");
  printf("  --mesh_col\tцвет линий сетки, [0.8 0.8 0.2];\n");
  printf("  --box_col\tцвет линий бокса, [1.0 1.0 1.0];\n");
  printf("  --sensor\tкоординаты сенсора, можно задавать несколько сенсоров;\n");
  return 0;
}
void read_float3(float* v, char* str) {
  for(int i=0; i<3; i++) { v[i] = strtof(str, &str); str++; }
}
float read_float(char* str) {
  return atof(str);
}
void add_sensor(int ix, int iy, int iz);

bool help_only=false, test_only=false;
int Tsteps=Ntime*10;
int _main(int argc, char** argv) {
  #ifdef MPI_ON
//  MPI_Init(&argc,&argv);
  int ismpith;
  MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &ismpith);
  switch(ismpith) {
    case MPI_THREAD_SINGLE:     printf("MPI multithreading implementation MPI_TREAD_SINGLE\n"); break;
    case MPI_THREAD_FUNNELED:   printf("MPI multithreading implementation MPI_TREAD_FUNNELED\n"); break;
    case MPI_THREAD_SERIALIZED: printf("MPI multithreading implementation MPI_THREAD_SERIALIZED\n"); break;
    case MPI_THREAD_MULTIPLE:   printf("MPI multithreading implementation MPI_THREAD_MULTIPLE\n"); break;
    default: printf("Unknown MPI multithreading implementation\n"); break;
  }
  //if (ismpith != MPI_THREAD_MULTIPLE) { printf("Error: MPI implementation does not support multithreading\n"); MPI_Abort(MPI_COMM_WORLD, 1); }
  MPI_Type_contiguous( sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, &MPI_DMDRAGTYPE );
  MPI_Type_contiguous( sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, &MPI_RAGPMLTYPE );
  MPI_Type_contiguous( sizeof(halfRag      )/sizeof(ftype), MPI_FTYPE, &MPI_HLFRAGTYPE );
  MPI_Type_commit(&MPI_DMDRAGTYPE);
  MPI_Type_commit(&MPI_RAGPMLTYPE);
  MPI_Type_commit(&MPI_HLFRAGTYPE);
  #endif
  argv ++; argc --;
  im3DHost.reset(); parsHost.swap_dir=new std::string("./");
  while(argc>0 && strncmp(*argv,"--",2)==0) {
    if(strncmp(*argv,"--help",6)==0) return print_help();
    else if(strcmp(*argv,"--test")==0) { test_only = true; argv ++; argc --; continue; }
    if(strcmp(*argv,"--box")==0) read_float3(im3DHost.BoxFactor, argv[1]);
    else if(strcmp(*argv,"--test")==0) test_only = true;
    else if(strcmp(*argv,"--mesh")==0) read_float3(im3DHost.MeshBox, argv[1]);
    else if(strcmp(*argv,"--Dmesh")==0) im3DHost.Dmesh=read_float(argv[1]);
    else if(strcmp(*argv,"--zoom")==0) read_float3(im3DHost.Dzoom, argv[1]);
    else if(strcmp(*argv,"--step")==0) read_float3(im3DHost.step, argv[1]);
    else if(strcmp(*argv,"--bkgr_col")==0) read_float3(im3DHost.bkgr_col, argv[1]);
    else if(strcmp(*argv,"--mesh_col")==0) read_float3(im3DHost.mesh_col, argv[1]);
    else if(strcmp(*argv,"--box_col")==0) read_float3(im3DHost.box_col, argv[1]);
    else if(strcmp(*argv,"--drop_dir")==0) strcpy(im3DHost.drop_dir,argv[1]);
    else if(strcmp(*argv,"--swap_dir")==0) parsHost.swap_dir=new std::string(argv[1]);
    else if(strcmp(*argv,"--sensor")==0) { float v[3]; read_float3(v, argv[1]); add_sensor(v[0], v[1], v[2]); }
    else { printf("Illegal parameters' syntax notation\n"); return print_help(); }
    //else if(strcmp(*argv,"--")==0) read_float3(im3DHost., argv[1]);
    printf("par: %s; vals: %s\n", argv[0], argv[1]);
    argv +=2; argc -=2;
  };
  im2D.get_device(2,0);
  if(test_only) printf("No GL\n");
  else printf("With GL\n");
try {
  if(type_diag_flag>=1) printf("Настройка опций визуализации по умолчанию\n");
  //imHost.reset();
  cudaTimer tm; tm.start();
  //if(GridNy>50) Tsteps=Ntime*10;
  parsHost.set();
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  copy2dev( parsHost, pars );
  copy2dev( shotpoint, src );
  shotpoint.check();
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  init();
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  init_index();
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  set_texture();
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  copy2dev( parsHost, pars );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );

  if(test_only) {
    for(int i=0; i<Tsteps/Ntime; i++) {
//    while(true) {
      tm.start();
      calcStep();
//      double tCpu=tm.stop();
//      printf("run time: %.2f msec, %.2f Gcells/sec\n", tCpu, 1.e-6*Ntime*Nx*Ny*Nz/tCpu);
//return 0;
    }
    return 0;
  }

  tm.start();
  parsHost.reset_im();
  im3DHost.reset(parsHost.arr4im);
  copy2dev( parsHost, pars );
  CreateShowTexModel();
  CHECK_ERROR( hipMemset(parsHost.arr4im.Arr3Dbuf,0,((long long int)Nx)*Ny*Nz*sizeof(ftype)) );
  mxw_draw<<<dim3((USE_UVM==2)?Np:Ns,Na),NT>>>(parsHost.arr4im.Arr3Dbuf);
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  im2D.get_device(2,0);
  im2D.init_image(argc,argv, im3DHost.bNx, im3DHost.bNy, "im3D");
  im3DHost.init3D(parsHost.arr4im); im3DHost.iz0=Nx-1; im3DHost.key_func('b',0,0);

  if(type_diag_flag>=1) printf("Настройка GLUT и запуск интерфейса\n");
  glutIdleFunc(idle_func);
  glutKeyboardFunc(key_func);
  glutMouseFunc(mouse_func);
  glutMotionFunc(motion_func);
  glutDisplayFunc(draw_func);
  if(type_diag_flag>=0) printf("Init cuda device: %.1f msec\n", tm.stop());
  glutMainLoop();
} catch(...) {
  printf("Возникла какая-то ошибка.\n");
}
  parsHost.clear();
  return -1;
}
int main(int argc, char** argv) {
  return _main(argc,argv);
}

float get_val_from_arr3D(int ix, int iy, int iz) {
  Arr3D_pars& arr=parsHost.arr4im;
  if(arr.inCPUmem) return arr.Arr3Dbuf[arr.get_ind(ix,iy,iz)];
  float res=0.0;
  if(arr.inGPUmem) exit_if_ERR(hipMemcpy(&res, arr.get_ptr(ix,iy,iz), sizeof(float), hipMemcpyDeviceToHost));
  return res;
}
Arr3D_pars& set_lim_from_arr3D() {
  Arr3D_pars& arr=parsHost.arr4im;
  if(arr.inCPUmem) arr.reset_min_max();
  if(arr.inGPUmem) {
    float* fLims=0,* fLimsD=0;
    exit_if_ERR(hipMalloc((void**) &fLimsD, 2*Ny*sizeof(float)));
    calc_limits<<<Ny,Nz>>>(arr.Arr3Dbuf, fLimsD);
    fLims=new float[2*Ny];
    exit_if_ERR(hipMemcpy(fLims, fLimsD, 2*Ny*sizeof(float), hipMemcpyDeviceToHost));
    exit_if_ERR(hipFree(fLimsD));
    arr.fMin = fLims[0]; arr.fMax = fLims[1];
    for(int i=0; i<Ny; i++) {
      if(fLims[2*i  ]<arr.fMin) arr.fMin = fLims[2*i  ];
      if(fLims[2*i+1]>arr.fMax) arr.fMax = fLims[2*i+1];
    }
    delete fLims;
  }
  return arr;
}
